#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hipfft/hipfft.h"

#include <stdio.h>

#include "CudaHelpLib.h"

static int isCudaOk = 0;
static _CUDA_DEV_INFO deviceinfo;

///dev constant define
__constant__  int devC_cols;
__constant__  int devC_rows;
__constant__  int devC_divc;
__constant__  int devC_x;
__constant__  int devC_y;
__constant__  float devC_f1;
__constant__  float devC_xe;
__constant__  float devC_ye;
__constant__  float devC_ze;
__constant__ unsigned int devC_Palette[512];
//float guassianTable[512];

////////////////////

static int *dev_temp_4M1 = 0;
static int *dev_temp_4M2 = 0;
static int *dev_temp_4M3 = 0;
static unsigned char *dev_background_4M = 0;
static unsigned char *dev_cuboid = 0;
////////////////////////////////



//>>>>>>>>>>>>>>>>share lib main func
#ifndef Q_OS_LINUX
#include "Windows.h"

BOOL APIENTRY DllMain(HMODULE hModule,
	DWORD  ul_reason_for_call,
	LPVOID lpReserved
	)
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
		break;
	case DLL_THREAD_ATTACH:
		break;
	case DLL_THREAD_DETACH:
		break;
	case DLL_PROCESS_DETACH:
		CuH_FreeTempCudaMem();
		break;
	}
	return TRUE;
}

#endif
//<<<<<<<<<<<<<


//////////////////////////////////////////////////////////////////////////
// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>cuda  kernel function //////////////////
//////////////////////////////////////////////////////////////////////

__global__ void magnitude32F_Kernel(FFT_Complex * datain, FFT_Real * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		float d = datain[index].re*datain[index].re + datain[index].im*datain[index].im;
		d = sqrtf(d);
		dataout[index] = d;
	}
}

__global__ void logAddBeta32F_Kernel(FFT_Real * datain, FFT_Real * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		float beta = *((float*)(&devC_y));
		float d = datain[index];
		d = d + beta;
		d = logf(d);
		dataout[index] = d;
	}
}

__global__ void cvtAndScale32Fto16U_Kernel(FFT_Real * datain, unsigned short * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		float d = *((float*)(&devC_x));
		float beta = *((float*)(&devC_y));
		d = d*datain[index] + beta;
		d = (d >= 0 && d <= 65535.0f)*d + (d > 65535.0f)*65535.0f;
		dataout[index] = d;
	}
}

__global__ void cpyRealToComplex_Kernel(FFT_Real * datain, FFT_Complex * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		dataout[index].re = datain[index];
		dataout[index].im = 0;
	}
}

__global__ void cpy16UC1ToComplex_Kernel(unsigned short * datain, FFT_Complex * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		dataout[index].re = datain[index];
		dataout[index].im = 0;
	}
}

__global__ void ROI_Complex_Kernel(FFT_Complex * datain, FFT_Complex *dataout) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		dataout[j*devC_cols + i].re = datain[(j + devC_y)*devC_divc + i + devC_x].re;
		dataout[j*devC_cols + i].im = datain[(j + devC_y)*devC_divc + i + devC_x].im;
	}
}

__global__ void transpose16UC1_Kernel(unsigned short * datain, unsigned short *dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned char data;
	if (i<devC_cols && j<devC_rows) {
		dataout[i*devC_rows + j] = datain[j*devC_cols + i];
	}
}

//////////////////////////////////////////////////////////////////////////
// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<cuda  kernel function //////////////////
//////////////////////////////////////////////////////////////////////


void CUDACALLMODE setIsCudaAccLibOK(int isok) {
	isCudaOk = isok;
}

int CUDACALLMODE getIsCudaAccLibOK(void) {
	return isCudaOk;
}

_CUDA_DEV_INFO* CUDACALLMODE getCudaDeviceInfo(int id) {
	hipFree(0);
	if (id >= 0 || id <= 4)
	{
		hipError_t res;
		hipDeviceProp_t device_prop;
		res = hipSetDevice(id);

		if (res != hipSuccess) {
			fprintf(stderr, "invaild cuda id!");
			return &deviceinfo;
		}
		device_prop.name[0] = 0;
		hipGetDeviceProperties(&device_prop, id);
		sprintf(deviceinfo.name, "%s", device_prop.name);
		deviceinfo.major = device_prop.major;
		deviceinfo.minor = device_prop.minor;
		deviceinfo.multiProcessorCount = device_prop.multiProcessorCount;
		deviceinfo.deviceOverlap = device_prop.deviceOverlap;
	}
	else {
		deviceinfo.name[0] = 0;
	}
	
	return &deviceinfo;
}

int getCudaDeviceCount(void) {
	int device_count;
	hipGetDeviceCount(&device_count);
	return device_count;
}

int  setCudaDevTo(int id) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(id);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

void CUDACALLMODE cudaNop(void) {
	hipError_t cudaStatus = hipSuccess;

	if (dev_temp_4M1 == 0) {
		cudaStatus = hipMalloc((void**)&dev_temp_4M1, 1024 * 1024 * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}

	if (dev_temp_4M2 == 0) {
		cudaStatus = hipMalloc((void**)&dev_temp_4M2, 1024 * 1024 * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}
	if (dev_temp_4M3 == 0) {
		cudaStatus = hipMalloc((void**)&dev_temp_4M3, 1024 * 1024 * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}
	if (dev_background_4M == 0) {
		cudaStatus = hipMalloc((void**)&dev_background_4M, 1024 * 1024 * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}
}

void CuH_FreeTempCudaMem(void) {
	if (dev_temp_4M1 != 0) {
		hipFree(dev_temp_4M1);
		dev_temp_4M1 = 0;
	}

	if (dev_temp_4M2 != 0) {
		hipFree(dev_temp_4M2);
		dev_temp_4M2 = 0;
	}

	if (dev_temp_4M3 != 0) {
		hipFree(dev_temp_4M2);
		dev_temp_4M3 = 0;
	}

	if (dev_background_4M != 0) {
		hipFree(dev_background_4M);
		dev_background_4M = 0;
	}


	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error!\n");
	}


	if (hipDeviceReset() != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
	}
}

int cudaDevSync(void) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int allocateFFTComplex(FFT_Complex** ptr, size_t size) {
	if (size < 1) {
		*ptr = nullptr;
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	FFT_Complex *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int allocateFFTComplexExt(FFT_Complex** ptr, int cols, int rows) {
	
	if (cols < 0) {
		*ptr = nullptr;
		return 0;
	}

	if (rows < 0) rows = 1;

	size_t size = static_cast<size_t>(cols)*static_cast<size_t>(rows)*sizeof(FFT_Complex);

	hipError_t cudaStatus = hipSuccess;
	FFT_Complex *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}

}

int allocateFFTReal(FFT_Real** ptr, size_t size) {
	if (size < 1) {
		*ptr = nullptr;
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	FFT_Real *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int allocateFFTRealExt(FFT_Real** ptr, int cols, int rows) {
	if (cols < 0) {
		*ptr = nullptr;
		return 0;
	}

	if (rows < 0) rows = 1;

	size_t size = static_cast<size_t>(cols)*static_cast<size_t>(rows)*sizeof(FFT_Real);

	hipError_t cudaStatus = hipSuccess;
	FFT_Real *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int freeCudaMem(void *ptr) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipFree(ptr);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int cudaMemFromHost(void *dstDev, void *srcHost, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstDev, srcHost, byteSize, hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}

}

int cudaMemToHost(void *dstHost, void *srcDev, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstHost, srcDev, byteSize, hipMemcpyDeviceToHost);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}

}

int cudaMemDevToDev(void *dstDev, void *srcDev, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstDev, srcDev, byteSize, hipMemcpyDeviceToDevice);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int destroyFFTPlan(FFTPlan_Handle plan) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftDestroy(plan);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int createFFTPlan1d_R2C(FFTPlan_Handle *plan, int cols, int rows) {
	if (cols < 0) {
		*plan = 0;
		return 0;
	}

	if (rows < 0) rows = 1;

	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftPlan1d(plan, cols, HIPFFT_R2C, rows);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int createFFTPlan1d_C2C(FFTPlan_Handle *plan, int cols, int rows) {
	if (cols < 0) {
		*plan = 0;
		return 0;
	}

	if (rows < 0) rows = 1;

	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftPlan1d(plan, cols, HIPFFT_C2C, rows);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int createFFTPlan1d_C2R(FFTPlan_Handle *plan, int cols, int rows) {
	if (cols < 0) {
		*plan = 0;
		return 0;
	}

	if (rows < 0) rows = 1;

	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftPlan1d(plan, cols, HIPFFT_C2R, rows);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}


int execR2CfftPlan(FFTPlan_Handle plan, FFT_Real *idata, FFT_Complex *odata) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftExecR2C(plan, idata, (hipfftComplex*)odata);
	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int execC2CfftPlan(FFTPlan_Handle plan, FFT_Complex *idata, FFT_Complex *odata ,int dir) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	int d = HIPFFT_FORWARD;
	if (d) {
		d = HIPFFT_BACKWARD;
	}
	cudaStatus = hipfftExecC2C(plan, (hipfftComplex*)idata, (hipfftComplex*)odata, d);
	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int execC2RfftPlan(FFTPlan_Handle plan, FFT_Complex *idata, FFT_Real *odata) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftExecC2R(plan,(hipfftComplex *) idata, (hipfftReal*)odata);
	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int CuH_magnitudeDevC2R(FFT_Complex *devSrc, int cols, int rows, FFT_Real *hostDst) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	FFT_Complex *srcPtr = 0;
	if (devSrc) {
		srcPtr = devSrc;
	}
	else {
		srcPtr = (FFT_Complex*)dev_temp_4M1;
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(FFT_Complex), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	magnitude32F_Kernel <<<gridS, blockS >>>(srcPtr, (FFT_Real*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "magnitude32F_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_magnitudeDevC2R returned error code %d after launching magnitude32F_Kernel!\n", cudaStatus);
		return 1;
	}

	if (hostDst) {
		cudaStatus = hipMemcpy(hostDst, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return 0;
}


int CuH_logDevR2R(FFT_Real *devSrc, int cols, int rows, float beta, FFT_Real *hostDst) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_y), &beta, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	FFT_Real *srcPtr = 0;
	if (devSrc) {
		srcPtr = devSrc;
	}
	else {
		srcPtr = (FFT_Real*)dev_temp_4M1;
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	logAddBeta32F_Kernel <<<gridS, blockS >>>(srcPtr, (FFT_Real*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "logAddBeta32F_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_logDevR2R returned error code %d after launching logAddBeta32F_Kernel!\n", cudaStatus);
		return 1;
	}

	if (hostDst) {
		cudaStatus = hipMemcpy(hostDst, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return 0;
}

int CuH_cvtDevRealTo16UC1(FFT_Real *devSrc, int cols, int rows, float alpha, float beta, unsigned short *hostDst) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_x), &alpha, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_y), &beta, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	FFT_Real *srcPtr = 0;
	if (devSrc) {
		srcPtr = devSrc;
	}
	else {
		srcPtr = (FFT_Real*)dev_temp_4M1;
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	cvtAndScale32Fto16U_Kernel <<<gridS, blockS >>>(srcPtr, (unsigned short*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cvtAndScale32Fto16U_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_cvtDevRealTo16UC1 returned error code %d after launching cvtAndScale32Fto16U_Kernel!\n", cudaStatus);
		return 1;
	}

	if (hostDst) {
		cudaStatus = hipMemcpy(hostDst, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return 0;
}


int CuH_cpyHostRealToDevComplex(FFT_Real *srcHost, FFT_Complex *dstDev, int cols, int rows) {
	if (!dstDev) return 1;
	
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	
	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	if (srcHost) {
		cudaStatus = hipMemcpy(dev_temp_4M1, srcHost, rows*cols*sizeof(FFT_Real), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	
	cpyRealToComplex_Kernel <<<gridS, blockS >>>((FFT_Real*)dev_temp_4M1, dstDev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cpyRealToComplex_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_cpyHostRealToDevComplex returned error code %d after launching cpyRealToComplex_Kernel!\n", cudaStatus);
		return 1;
	}

	
		//cudaStatus = hipMemcpy(dev_temp_4M2, dstDev, rows*cols*sizeof(FFT_Complex), hipMemcpyDeviceToDevice);
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipMemcpy failed!");
		//	return 1;
		//}


	return 0;

}

int CuH_cpy16UC1ToDevComplex(unsigned short *srcHost, FFT_Complex *dstDev, int cols, int rows) {
	if (!dstDev) return 1;

	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}


	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	if (srcHost) {
		cudaStatus = hipMemcpy(dev_temp_4M1, srcHost, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	cpy16UC1ToComplex_Kernel <<<gridS, blockS >>>((unsigned short*)dev_temp_4M1, dstDev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cpy16UC1ToComplex_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_cpyHostRealToDevComplex returned error code %d after launching cpy16UC1ToComplex_Kernel!\n", cudaStatus);
		return 1;
	}


	//cudaStatus = hipMemcpy(dev_temp_4M2, dstDev, rows*cols*sizeof(FFT_Complex), hipMemcpyDeviceToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	return 1;
	//}


	return 0;
}

int CuH_ROIdevComplex(FFT_Complex *dataDev, int cols, int rows, int x, int y, int width, int height) {

	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_x), &x, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_y), &y, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &width, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &height, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_divc), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	FFT_Complex *srcCptr = nullptr;
	if (dataDev) {
		srcCptr = dataDev;
	}
	else {
		srcCptr = (FFT_Complex *)dev_temp_4M2;
	}
	cudaStatus = hipMemcpy((void*)dev_temp_4M1, (void*)srcCptr, rows*cols*sizeof(FFT_Complex), hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}


	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(width / 16, height / 16);
	if (width % 16) {
		gridS.x += 1;
	}
	if (height % 16) {
		gridS.y += 1;
	}

	ROI_Complex_Kernel<<<gridS, blockS >>>((FFT_Complex*)dev_temp_4M1, srcCptr);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ROI_Complex_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching ROI_Complex_Kernel!\n", cudaStatus);
		return 1;
	}

	return  0;

}

int  transpose16UC1(int rows, int cols, void* dev_src, void *dev_dst)
{
	int res = 0;
	hipError_t cudaStatus = hipSuccess;
	//copy constant

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}
	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}


	//invoke kernel
	transpose16UC1_Kernel <<<gridS, blockS >>>((unsigned short *)dev_src, (unsigned short *)dev_dst);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "transpose16UC1 Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transpose16UC1_Kernel!\n", cudaStatus);
		return 1;
	}

	return res;
}

int CuH_transpose16UC1(int rows, int cols, void* host_src, void *host_dst) {
	hipError_t cudaStatus = hipSuccess;

	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	if (host_src) {
		cudaStatus = hipMemcpy(dev_temp_4M1, host_src, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	if (transpose16UC1(rows, cols, (void*)dev_temp_4M1, (void*)dev_temp_4M2)) return 1;

	if (host_dst) {
		cudaStatus = hipMemcpy(host_dst, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return  0;
}