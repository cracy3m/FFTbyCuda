
#include "hip/hip_runtime.h"
#include ""
#include "hipfft/hipfft.h"

#include <stdio.h>

#include "CudaHelpLib.h"

static int isCudaOk = 0;
static _CUDA_DEV_INFO deviceinfo;

void CUDACALLMODE setIsCudaAccLibOK(int isok) {
	isCudaOk = isok;
}

int CUDACALLMODE getIsCudaAccLibOK(void) {
	return isCudaOk;
}

_CUDA_DEV_INFO* CUDACALLMODE getCudaDeviceInfo(int id) {
	hipFree(0);
	if (id >= 0 || id <= 4)
	{
		hipError_t res;
		hipDeviceProp_t device_prop;
		res = hipSetDevice(id);

		if (res != hipSuccess) {
			fprintf(stderr, "invaild cuda id!");
			return &deviceinfo;
		}
		device_prop.name[0] = 0;
		hipGetDeviceProperties(&device_prop, id);
		sprintf(deviceinfo.name, "%s", device_prop.name);
		deviceinfo.major = device_prop.major;
		deviceinfo.minor = device_prop.minor;
		deviceinfo.multiProcessorCount = device_prop.multiProcessorCount;
		deviceinfo.deviceOverlap = device_prop.deviceOverlap;
	}
	else {
		deviceinfo.name[0] = 0;
	}
	
	return &deviceinfo;
}

int getCudaDeviceCount(void) {
	int device_count;
	hipGetDeviceCount(&device_count);
	return device_count;
}

int  setCudaDevTo(int id) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(id);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int cudaDevSync(void) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int allocateFFTComplex(FFT_Complex** ptr, size_t size) {
	if (size < 1) {
		*ptr = nullptr;
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	FFT_Complex *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int allocateFFTComplexExt(FFT_Complex** ptr, int cols, int rows) {
	
	if (cols < 0) {
		*ptr = nullptr;
		return 0;
	}

	if (rows < 0) rows = 1;

	size_t size = static_cast<size_t>(cols)*static_cast<size_t>(rows)*sizeof(FFT_Complex);

	hipError_t cudaStatus = hipSuccess;
	FFT_Complex *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}

}

int allocateFFTReal(FFT_Real** ptr, size_t size) {
	if (size < 1) {
		*ptr = nullptr;
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	FFT_Real *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int allocateFFTRealExt(FFT_Real** ptr, int cols, int rows) {
	if (cols < 0) {
		*ptr = nullptr;
		return 0;
	}

	if (rows < 0) rows = 1;

	size_t size = static_cast<size_t>(cols)*static_cast<size_t>(rows)*sizeof(FFT_Real);

	hipError_t cudaStatus = hipSuccess;
	FFT_Real *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int freeCudaMem(void *ptr) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipFree(ptr);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int cudaMemFromHost(void *dstDev, void *srcHost, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstDev, srcHost, byteSize, hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}

}

int cudaMemToHost(void *dstHost, void *srcDev, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstHost, srcDev, byteSize, hipMemcpyDeviceToHost);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}

}

int cudaMemDevToDev(void *dstDev, void *srcDev, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstDev, srcDev, byteSize, hipMemcpyDeviceToDevice);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int createFFTPlan1d_R2C(FFTPlan_Handle *plan, int cols, int rows) {
	if (cols < 0) {
		*plan = 0;
		return 0;
	}

	if (rows < 0) rows = 1;

	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftPlan1d(plan, cols, HIPFFT_R2C, rows);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int destroyFFTPlan(FFTPlan_Handle plan) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftDestroy(plan);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int execR2CfftPlan(FFTPlan_Handle plan, FFT_Real *idata, FFT_Complex *odata) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftExecR2C(plan, idata, (hipfftComplex*)odata);
	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}