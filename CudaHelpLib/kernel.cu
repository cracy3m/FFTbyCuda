#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hipfft/hipfft.h"

#include <stdio.h>

#include "CudaHelpLib.h"

static int isCudaOk = 0;
static _CUDA_DEV_INFO deviceinfo;

///dev constant define
__constant__  int devC_cols;
__constant__  int devC_rows;
__constant__  int devC_divc;
__constant__  int devC_x;
__constant__  int devC_y;
__constant__  float devC_f1;
__constant__  float devC_xe;
__constant__  float devC_ye;
__constant__  float devC_ze;
__constant__ unsigned int devC_Palette[512];
//float guassianTable[512];

////////////////////

static int *dev_temp_4M1 = 0;
static int *dev_temp_4M2 = 0;
static int *dev_temp_4M3 = 0;
static unsigned char *dev_background_4M = 0;
static unsigned char *dev_cuboid = 0;
////////////////////////////////



//>>>>>>>>>>>>>>>>share lib main func
#ifndef Q_OS_LINUX
#include "Windows.h"

BOOL APIENTRY DllMain(HMODULE hModule,
	DWORD  ul_reason_for_call,
	LPVOID lpReserved
	)
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
		break;
	case DLL_THREAD_ATTACH:
		break;
	case DLL_THREAD_DETACH:
		break;
	case DLL_PROCESS_DETACH:
		CuH_FreeTempCudaMem();
		break;
	}
	return TRUE;
}

#endif
//<<<<<<<<<<<<<


//////////////////////////////////////////////////////////////////////////
// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>cuda  kernel function //////////////////
//////////////////////////////////////////////////////////////////////

__global__ void magnitude32F_Kernel(FFT_Complex * datain, FFT_Real * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		float d = datain[index].re*datain[index].re + datain[index].im*datain[index].im;
		d = sqrtf(d);
		dataout[index] = d;
	}
}

__global__ void logAddBeta32F_Kernel(FFT_Real * datain, FFT_Real * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		float beta = *((float*)(&devC_y));
		float d = datain[index];
		d = d + beta;
		d = logf(d);
		dataout[index] = d;
	}
}

__global__ void cvtAndScale32Fto16U_Kernel(FFT_Real * datain, unsigned short * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		float d = *((float*)(&devC_x));
		float beta = *((float*)(&devC_y));
		d = d*datain[index] + beta;
		d = (d >= 0 && d <= 65535.0f)*d + (d > 65535.0f)*65535.0f;
		dataout[index] = d;
	}
}

__global__ void cpyRealToComplex_Kernel(FFT_Real * datain, FFT_Complex * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		dataout[index].re = datain[index];
		dataout[index].im = 0;
	}
}

__global__ void cpy16UC1ToComplex_Kernel(unsigned short * datain, FFT_Complex * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		dataout[index].re = datain[index];
		dataout[index].im = 0;
	}
}

__global__ void ROI_Complex_Kernel(FFT_Complex * datain, FFT_Complex *dataout) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		dataout[j*devC_cols + i].re = datain[(j + devC_y)*devC_divc + i + devC_x].re;
		dataout[j*devC_cols + i].im = datain[(j + devC_y)*devC_divc + i + devC_x].im;
	}
}

__global__ void transposeComplex_Kernel(FFT_Complex * datain, FFT_Complex *dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	//unsigned char data;
	if (i<devC_cols && j<devC_rows) {
		dataout[i*devC_rows + j].re = datain[j*devC_cols + i].re;
		dataout[i*devC_rows + j].im = datain[j*devC_cols + i].im;
	}
}

__global__ void transpose32FC1_Kernel(float * datain, float *dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	//unsigned char data;
	if (i<devC_cols && j<devC_rows) {
		dataout[i*devC_rows + j] = datain[j*devC_cols + i];
	}
}

__global__ void transpose16UC1_Kernel(unsigned short * datain, unsigned short *dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	//unsigned char data;
	if (i<devC_cols && j<devC_rows) {
		dataout[i*devC_rows + j] = datain[j*devC_cols + i];
	}
}

__global__ void calcWinAndDispersion(FFT_Complex *data, FFT_Real *wind, FFT_Complex *dispersion) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		const int index = j*devC_cols + i;
		FFT_Real d1 = data[index].re*dispersion[i].re - data[index].im*dispersion[i].im;
		FFT_Real d2 = data[index].im*dispersion[i].re + data[index].re*dispersion[i].im;
		d1 *= wind[i];
		d2 *= wind[i];
		data[index].re = d1;
		data[index].im = d2;
	}
}

__global__ void power8UC1_Kernel(unsigned char * datain, unsigned char * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		float d = datain[j*devC_cols + i];
		d = powf(d, devC_f1);
		if (d>255.0f) d = 255.0f;
		dataout[j*devC_cols + i] = d;
	}
}

__global__ void pixWindow16UC1To8UC1_Kernel(unsigned short * datain, unsigned char * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	float s_v = devC_y;
	float delta = devC_y;
	s_v /= 2.0f;
	s_v = devC_x - s_v;
	delta = 256.0f / delta;

	if (i<devC_cols && j<devC_rows) {
		float d = datain[j*devC_cols + i];
		d = d - s_v;
		d = d*delta;
		d = d*(d>0.0f);
		d = d*(d <= 255.0f);    //+255.0*(d>255.0f)
		dataout[j*devC_cols + i] = d;
	}
}

__global__ void allPixAvg_Kernel(unsigned short *datain , float *dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int index = j * 256 + i;
	float sum = 0;

	if (index < devC_cols) {
		for (int k = 0; k < devC_rows; k++)
		{
			sum = sum + datain[k*devC_cols + index];
			if (k > 0) sum /= 2.0f;
		}
		dataout[index] = sum;
	}
	
}

__global__ void threshold16UC1_Kernel(unsigned short * datain, unsigned short * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		int th = 0;
		th = (datain[j*devC_cols + i] >= devC_x);
		if (devC_divc & 0x0080) th = ~th;
		if ((devC_divc & 0x000F) == 0) {
			dataout[j*devC_cols + i] = th * 65535;
		}
		else if ((devC_divc & 0x000F) == 1) {
			dataout[j*devC_cols + i] = th*datain[j*devC_cols + i];
		}

	}

}

__global__ void zeroLeftComplexAndDivConst_Kernel(FFT_Complex * data) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < devC_cols && j < devC_rows) {
		const int index = j*devC_cols + i;
		data[index].re = (data[index].re / devC_f1)*(i >= (devC_cols/2));
		data[index].im = (data[index].im / devC_f1)*(i >= (devC_cols/2));
	}
}


__global__ void zeroComplexReOrIm_Kernel(FFT_Complex * data) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < devC_cols && j < devC_rows) {
		const int index = j*devC_cols + i;
		data[index].re = (data[index].re )*(devC_divc == 0);
		data[index].im = (data[index].im)*(devC_divc == 1);
	}
}


__global__ void flipH8UC1_Kernel(unsigned char * datain, unsigned char * dataout) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i<devC_cols && j<devC_rows) {
		dataout[j*devC_cols + i] = datain[(devC_rows - 1 - j)*devC_cols + i];
	}

}


//////////////////////////////////////////////////////////////////////////
// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<cuda  kernel function //////////////////
//////////////////////////////////////////////////////////////////////


void CUDACALLMODE setIsCudaAccLibOK(int isok) {
	isCudaOk = isok;
}

int CUDACALLMODE getIsCudaAccLibOK(void) {
	return isCudaOk;
}

_CUDA_DEV_INFO* CUDACALLMODE getCudaDeviceInfo(int id) {
	hipFree(0);
	if (id >= 0 || id <= 4)
	{
		hipError_t res;
		hipDeviceProp_t device_prop;
		res = hipSetDevice(id);

		if (res != hipSuccess) {
			fprintf(stderr, "invaild cuda id!");
			return &deviceinfo;
		}
		device_prop.name[0] = 0;
		hipGetDeviceProperties(&device_prop, id);
		sprintf(deviceinfo.name, "%s", device_prop.name);
		deviceinfo.major = device_prop.major;
		deviceinfo.minor = device_prop.minor;
		deviceinfo.multiProcessorCount = device_prop.multiProcessorCount;
		deviceinfo.deviceOverlap = device_prop.deviceOverlap;
	}
	else {
		deviceinfo.name[0] = 0;
	}
	
	return &deviceinfo;
}

int getCudaDeviceCount(void) {
	int device_count;
	hipGetDeviceCount(&device_count);
	return device_count;
}

int  setCudaDevTo(int id) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(id);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

void CUDACALLMODE cudaNop(void) {
	hipError_t cudaStatus = hipSuccess;

	if (dev_temp_4M1 == 0) {
		cudaStatus = hipMalloc((void**)&dev_temp_4M1, 1024 * 1024 * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}

	if (dev_temp_4M2 == 0) {
		cudaStatus = hipMalloc((void**)&dev_temp_4M2, 1024 * 1024 * 64);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}
	if (dev_temp_4M3 == 0) {
		cudaStatus = hipMalloc((void**)&dev_temp_4M3, 1024 * 1024 * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}
	if (dev_background_4M == 0) {
		cudaStatus = hipMalloc((void**)&dev_background_4M, 1024 * 1024 * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");

		}
	}
}

void CuH_FreeTempCudaMem(void) {
	if (dev_temp_4M1 != 0) {
		hipFree(dev_temp_4M1);
		dev_temp_4M1 = 0;
	}

	if (dev_temp_4M2 != 0) {
		hipFree(dev_temp_4M2);
		dev_temp_4M2 = 0;
	}

	if (dev_temp_4M3 != 0) {
		hipFree(dev_temp_4M2);
		dev_temp_4M3 = 0;
	}

	if (dev_background_4M != 0) {
		hipFree(dev_background_4M);
		dev_background_4M = 0;
	}


	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error!\n");
	}


	if (hipDeviceReset() != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
	}
}

int cudaDevSync(void) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int allocateFFTComplex(FFT_Complex** ptr, size_t size) {
	if (size < 1) {
		*ptr = nullptr;
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	FFT_Complex *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int allocateFFTComplexExt(FFT_Complex** ptr, int cols, int rows) {
	
	if (cols < 0) {
		*ptr = nullptr;
		return 0;
	}

	if (rows < 0) rows = 1;

	size_t size = static_cast<size_t>(cols)*static_cast<size_t>(rows)*sizeof(FFT_Complex);

	hipError_t cudaStatus = hipSuccess;
	FFT_Complex *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}

}

int allocateFFTReal(FFT_Real** ptr, size_t size) {
	if (size < 1) {
		*ptr = nullptr;
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	FFT_Real *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int allocateFFTRealExt(FFT_Real** ptr, int cols, int rows) {
	if (cols < 0) {
		*ptr = nullptr;
		return 0;
	}

	if (rows < 0) rows = 1;

	size_t size = static_cast<size_t>(cols)*static_cast<size_t>(rows)*sizeof(FFT_Real);

	hipError_t cudaStatus = hipSuccess;
	FFT_Real *res = nullptr;
	cudaStatus = hipMalloc((void **)&res, size);
	if (cudaStatus == hipSuccess && res) {
		*ptr = res;
		return 1;
	}
	else {
		*ptr = nullptr;
		return 0;
	}
}

int freeCudaMem(void *ptr) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipFree(ptr);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int cudaMemFromHost(void *dstDev, void *srcHost, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstDev, srcHost, byteSize, hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}

}

int cudaMemToHost(void *dstHost, void *srcDev, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstHost, srcDev, byteSize, hipMemcpyDeviceToHost);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}

}

int cudaMemDevToDev(void *dstDev, void *srcDev, size_t byteSize) {
	if (byteSize < 1) {
		return 0;
	}

	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dstDev, srcDev, byteSize, hipMemcpyDeviceToDevice);
	if (cudaStatus == hipSuccess) {
		return 1;
	}
	else {
		return 0;
	}
}

int destroyFFTPlan(FFTPlan_Handle plan) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftDestroy(plan);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int createFFTPlan1d_R2C(FFTPlan_Handle *plan, int cols, int rows) {
	if (cols < 0) {
		*plan = 0;
		return 0;
	}

	if (rows < 0) rows = 1;

	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftPlan1d(plan, cols, HIPFFT_R2C, rows);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int createFFTPlan1d_C2C(FFTPlan_Handle *plan, int cols, int rows) {
	if (cols < 0) {
		*plan = 0;
		return 0;
	}

	if (rows < 0) rows = 1;

	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftPlan1d(plan, cols, HIPFFT_C2C, rows);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int createFFTPlan1d_C2R(FFTPlan_Handle *plan, int cols, int rows) {
	if (cols < 0) {
		*plan = 0;
		return 0;
	}

	if (rows < 0) rows = 1;

	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftPlan1d(plan, cols, HIPFFT_C2R, rows);

	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}


int execR2CfftPlan(FFTPlan_Handle plan, FFT_Real *idata, FFT_Complex *odata) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftExecR2C(plan, idata, (hipfftComplex*)odata);
	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int execC2CfftPlan(FFTPlan_Handle plan, FFT_Complex *idata, FFT_Complex *odata ,int dir) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	int d = HIPFFT_FORWARD;
	if (dir) {
		d = HIPFFT_BACKWARD;
	}
	cudaStatus = hipfftExecC2C(plan, (hipfftComplex*)idata, (hipfftComplex*)odata, d);
	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}

int execC2RfftPlan(FFTPlan_Handle plan, FFT_Complex *idata, FFT_Real *odata) {
	hipfftResult cudaStatus = HIPFFT_SUCCESS;
	cudaStatus = hipfftExecC2R(plan,(hipfftComplex *) idata, (hipfftReal*)odata);
	if (cudaStatus == HIPFFT_SUCCESS) {
		return 1;
	}
	else {
		return 0;
	}
}


int CuH_downloadTemp4M2(int size, unsigned char* host_dst) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy((void*)host_dst, dev_temp_4M2, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}

	return 0;
}


int CuH_uploadTemp4M2(int size, unsigned char* host_dst) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(dev_temp_4M2, (void*)host_dst, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}

	return 0;
}


int CuH_magnitudeDevC2R(FFT_Complex *devSrc, int cols, int rows, FFT_Real *hostDst) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	FFT_Complex *srcPtr = 0;
	if (devSrc) {
		srcPtr = devSrc;
	}
	else {
		srcPtr = (FFT_Complex*)dev_temp_4M2;
	}

	magnitude32F_Kernel <<<gridS, blockS >>>(srcPtr, (FFT_Real*)dev_temp_4M1);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "magnitude32F_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_magnitudeDevC2R returned error code %d after launching magnitude32F_Kernel!\n", cudaStatus);
		return 1;
	}

	if (hostDst) {
		cudaStatus = hipMemcpy(hostDst, dev_temp_4M1, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy(dev_temp_4M2, dev_temp_4M1, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return 0;
}


int CuH_logDevR2R(FFT_Real *devSrc, int cols, int rows, float beta, FFT_Real *hostDst) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_y), &beta, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	FFT_Real *srcPtr = 0;
	if (devSrc) {
		srcPtr = devSrc;
	}
	else {
		srcPtr = (FFT_Real*)dev_temp_4M1;
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	logAddBeta32F_Kernel <<<gridS, blockS >>>(srcPtr, (FFT_Real*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "logAddBeta32F_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_logDevR2R returned error code %d after launching logAddBeta32F_Kernel!\n", cudaStatus);
		return 1;
	}

	if (hostDst) {
		cudaStatus = hipMemcpy(hostDst, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return 0;
}

int CuH_cvtDevRealTo16UC1(FFT_Real *devSrc, int cols, int rows, float alpha, float beta, unsigned short *hostDst) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_x), &alpha, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_y), &beta, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	FFT_Real *srcPtr = 0;
	if (devSrc) {
		srcPtr = devSrc;
	}
	else {
		srcPtr = (FFT_Real*)dev_temp_4M1;
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	cvtAndScale32Fto16U_Kernel <<<gridS, blockS >>>(srcPtr, (unsigned short*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cvtAndScale32Fto16U_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_cvtDevRealTo16UC1 returned error code %d after launching cvtAndScale32Fto16U_Kernel!\n", cudaStatus);
		return 1;
	}

	if (hostDst) {
		cudaStatus = hipMemcpy(hostDst, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return 0;
}


int CuH_cpyHostRealToDevComplex(FFT_Real *srcHost, FFT_Complex *dstDev, int cols, int rows) {
	if (!dstDev) return 1;
	
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	
	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	if (srcHost) {
		cudaStatus = hipMemcpy(dev_temp_4M1, srcHost, rows*cols*sizeof(FFT_Real), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(FFT_Real), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	
	cpyRealToComplex_Kernel <<<gridS, blockS >>>((FFT_Real*)dev_temp_4M1, dstDev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cpyRealToComplex_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_cpyHostRealToDevComplex returned error code %d after launching cpyRealToComplex_Kernel!\n", cudaStatus);
		return 1;
	}

	
		//cudaStatus = hipMemcpy(dev_temp_4M2, dstDev, rows*cols*sizeof(FFT_Complex), hipMemcpyDeviceToDevice);
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipMemcpy failed!");
		//	return 1;
		//}


	return 0;

}

int CuH_cpy16UC1ToDevComplex(unsigned short *srcHost, FFT_Complex *dstDev, int cols, int rows) {
	if (!dstDev) return 1;

	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}


	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	if (srcHost) {
		cudaStatus = hipMemcpy(dev_temp_4M1, srcHost, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	cpy16UC1ToComplex_Kernel <<<gridS, blockS >>>((unsigned short*)dev_temp_4M1, dstDev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cpy16UC1ToComplex_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_cpyHostRealToDevComplex returned error code %d after launching cpy16UC1ToComplex_Kernel!\n", cudaStatus);
		return 1;
	}


	//cudaStatus = hipMemcpy(dev_temp_4M2, dstDev, rows*cols*sizeof(FFT_Complex), hipMemcpyDeviceToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	return 1;
	//}


	return 0;
}

int CuH_ROIdevComplex(FFT_Complex *dataDev, int cols, int rows, int x, int y, int width, int height) {

	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_x), &x, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_y), &y, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &width, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &height, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_divc), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(width / 16, height / 16);
	if (width % 16) {
		gridS.x += 1;
	}
	if (height % 16) {
		gridS.y += 1;
	}

	ROI_Complex_Kernel<<<gridS, blockS >>>(dataDev, (FFT_Complex *)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ROI_Complex_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching ROI_Complex_Kernel!\n", cudaStatus);
		return 1;
	}

	cudaStatus = hipMemcpy(dataDev, dev_temp_4M2, width*height*sizeof(FFT_Complex), hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}

	return  0;

}

int  transpose16UC1(int rows, int cols, void* dev_src, void *dev_dst)
{
	int res = 0;
	hipError_t cudaStatus = hipSuccess;
	//copy constant

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}
	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}


	//invoke kernel
	transpose16UC1_Kernel <<<gridS, blockS >>>((unsigned short *)dev_src, (unsigned short *)dev_dst);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "transpose16UC1 Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transpose16UC1_Kernel!\n", cudaStatus);
		return 1;
	}

	return res;
}

int CuH_transposeComplex(int rows, int cols, FFT_Complex* dev_src, FFT_Complex *dev_dst) {
	hipError_t cudaStatus = hipSuccess;

	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	//invoke kernel
	transposeComplex_Kernel <<<gridS, blockS >>>(dev_src, dev_dst);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "transposeComplex_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transposeComplex_Kernel!\n", cudaStatus);
		return 1;
	}

	return 0;
}

int CuH_tranDevCtoHostDouble(int rows, int cols, FFT_Complex* dev_src, double *host_dst) {
	hipError_t cudaStatus = hipSuccess;

	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}
}

int CuH_transpose32FC1(int rows, int cols, void* dev_src, void *dev_dst) {
	hipError_t cudaStatus = hipSuccess;

	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	float *datain = nullptr;

	if (dev_src) {
		datain = (float *)dev_src;
	}
	else {
		datain = (float *)dev_temp_4M1;
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(float), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	float *dataout = nullptr;
	if (dev_dst) {
		dataout = (float *)dev_dst;
	}
	else {
		dataout = (float *)dev_temp_4M2;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}


	//invoke kernel
	transpose32FC1_Kernel <<<gridS, blockS >>>(datain, dataout);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "transpose32FC1_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transpose32FC1_Kernel!\n", cudaStatus);
		return 1;
	}

	return 0;
}

int CuH_transpose16UC1(int rows, int cols, void* host_src, void *host_dst) {
	hipError_t cudaStatus = hipSuccess;

	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	if (host_src) {
		cudaStatus = hipMemcpy(dev_temp_4M1, host_src, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	if (transpose16UC1(rows, cols, (void*)dev_temp_4M1, (void*)dev_temp_4M2)) return 1;

	if (host_dst) {
		cudaStatus = hipMemcpy(host_dst, dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	return  0;
}

int CuH_devCdataCalcWinAndDispersion(int cols, int rows, FFT_Complex *dataDev, FFT_Real *winDev, FFT_Complex *dispersionDev) {

	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}


	calcWinAndDispersion <<<gridS, blockS >>>(dataDev, winDev, dispersionDev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "calcWinAndDispersion Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CuH_devCdataCalcWinAndDispersion returned error code %d after launching calcWinAndDispersion!\n", cudaStatus);
		return 1;
	}

	return 0;
}


int CuH_power8UC1(int rows, int cols, float p) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_f1), &p, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols, hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	power8UC1_Kernel <<<gridS, blockS >>>((unsigned char*)dev_temp_4M1, (unsigned char*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "power8UC1 Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching power8UC1_Kernel!\n", cudaStatus);
		return 1;
	}

	return 0;
}


int CuH_pixWindow16UC1To8UC1(int rows, int cols, int winCenter, int winWidth, unsigned short *host_src) {
	
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_x), &winCenter, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_y), &winWidth, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	if (!host_src) {
		cudaStatus = hipMemcpy((void*)dev_temp_4M1, (void*)dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy((void*)dev_temp_4M1, (void*)host_src, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	pixWindow16UC1To8UC1_Kernel <<<gridS, blockS >>>((unsigned short*)dev_temp_4M1, (unsigned char*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "pixWindow16UC1To8UC1_Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pixWindow16UC1To8UC1_Kernel!\n", cudaStatus);
		return 1;
	}
	
	return 0;

}


int CuH_allPixAvgValue(int rows, int cols, unsigned short* host_src, float *host_res) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0 || dev_temp_4M3 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	if (!host_src) {
		cudaStatus = hipMemcpy((void*)dev_temp_4M1, (void*)dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy((void*)dev_temp_4M1, (void*)host_src, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}


	//calc block size
	int dimx = 256;
	int dimy = cols / dimx;
	if (cols % dimx) {
		dimy += 1;
	}

	allPixAvg_Kernel <<<dimy, dimx >>>((unsigned short*)dev_temp_4M1, (float *)dev_temp_4M3);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "allPixAvg_Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching allPixAvg_Kernel!\n", cudaStatus);
		return 1;
	}

	float *avgArr = (float*)malloc(rows*cols*sizeof(float));
	if (!avgArr) {
		fprintf(stderr, "malloc() failed!");
		return 1;
	}

	cudaStatus = hipMemcpy((void*)avgArr, (void*)dev_temp_4M3, cols*sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		if (avgArr) {
			free(avgArr);
			avgArr = nullptr;
		}
		return 1;
	}

	for (int i = 1; i < cols; i++)
	{
		avgArr[0] += avgArr[i];
		avgArr[0] /= 2.0f;
	}
	
	host_res[0] = avgArr[0];

	if (avgArr) {
		free(avgArr);
		avgArr = nullptr;
	}
	
	return 0;
}

int CuH_threshold16UC1(int rows, int cols, int thres, int mode, unsigned short* host_src) {
	
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_x), &thres, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_divc), &mode, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	if (host_src) {
		cudaStatus = hipMemcpy((void*)dev_temp_4M1, (void*)host_src, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	else {
		cudaStatus = hipMemcpy((void*)dev_temp_4M1, (void*)dev_temp_4M2, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}
	}
	
	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}
	
	threshold16UC1_Kernel <<<gridS, blockS >>>((unsigned short*)dev_temp_4M1, (unsigned short*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "threshold16UC1_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching threshold16UC1_Kernel!\n", cudaStatus);
		return 1;
	}
	
	return 0;
	
}


int CuH_zeroLeftComplexAndDivConst(int rows, int cols, float divConst, FFT_Complex *dataDev) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_f1), &divConst, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	zeroLeftComplexAndDivConst_Kernel <<<gridS, blockS >>>(dataDev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "zeroLeftComplexAndDivConst_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching zeroLeftComplexAndDivConst_Kernel!\n", cudaStatus);
		return 1;
	}

	return 0;
}


int CuH_zeroComplexReOrIm(int rows, int cols, int mode, FFT_Complex *dataDev) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_divc), &mode, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	zeroComplexReOrIm_Kernel <<<gridS, blockS >>>(dataDev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "zeroComplexReOrIm_Kernel Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching zeroComplexReOrIm_Kernel!\n", cudaStatus);
		return 1;
	}

	return 0;
}


int CuH_flipH8UC1(int rows, int cols) {
	hipError_t cudaStatus = hipSuccess;
	if (dev_temp_4M1 == 0 || dev_temp_4M2 == 0) {
		printf("cuda mem alloc faild.\n");
		return 1;
	}


	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_rows), &rows, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(devC_cols), &cols, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "set const var failed!");
		return 1;
	}

	cudaStatus = hipMemcpy(dev_temp_4M1, dev_temp_4M2, rows*cols, hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}

	//calc block size
	dim3 blockS(16, 16);
	dim3 gridS(cols / 16, rows / 16);
	if (cols % 16) {
		gridS.x += 1;
	}
	if (rows % 16) {
		gridS.y += 1;
	}

	flipH8UC1_Kernel <<<gridS, blockS >>>((unsigned char*)dev_temp_4M1, (unsigned char*)dev_temp_4M2);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "flipH8UC1 Kernel failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//wait kernel finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching flipH8UC1_Kernel!\n", cudaStatus);
		return 1;
	}


	return  0;
}